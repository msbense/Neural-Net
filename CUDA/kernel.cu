#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C"
{
	__global__ void aKernal(int *a)
	{
		int i = threadIdx.x;
		a[i] = i;
	}
}

int main()
{
	return 0;
}