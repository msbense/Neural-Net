#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <device_atomic_functions.h>
#include <device_double_functions.h>
#include <math.h>

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C"
{
	//Multiplies input vector by weight vector
	__global__ void FeedFoward(int *inputs, int *weightMatrix, int *activations, int numInputNeurons)
	{
		int neuronIdx = blockIdx.x;
		int inputNeuronIdx = threadIdx.x;
		__shared__ double sum;
		double result = weightMatrix[(neuronIdx * blockDim.x) + inputNeuronIdx] * inputs[inputNeuronIdx];
		
		//atomicAdd using doubles
		unsigned long long int* address_as_ull = (unsigned long long int*) &sum;
		unsigned long long int old = *address_as_ull, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(result +
				__longlong_as_double(assumed)));
		} while (assumed != old);
		__syncthreads();

		if (inputNeuronIdx == 0) 
		{
			activations[neuronIdx] = (1/(1+exp(-sum)));
		}
	}

	
}
int main()
{
	return 0;
}