#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <device_atomic_functions.h>
#include <device_double_functions.h>
#include <math.h>

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C"
{
	//Multiplies input vector by weight vector
	__global__ void FeedFoward(double *inputs, double *weightMatrix, double *activations, int numInputNeurons)
	{
		int neuronIdx = threadIdx.x;
		double sum;
		for (int inputNeuronIdx = 0; inputNeuronIdx < numInputNeurons; inputNeuronIdx++)
		{
			sum += weightMatrix[(neuronIdx * numInputNeurons) + inputNeuronIdx] * inputs[inputNeuronIdx];
		}
		activations[neuronIdx] = (1/(1+exp(-sum)));
	}

	__global__ void BackPropFirstLayer(double* errors, double* activations, double* correct) 
	{
		int neuronIdx = threadIdx.x;
		errors[neuronIdx] = activations[neuronIdx] - correct[neuronIdx];
	}

	__global__ void BackProp(double* input_errors, double* output_errors, double* activations, double* weightMatrix, int numOutputNeurons, int numInputNeurons)
	{
		int neuronIdx = threadIdx.x;
		double sum = 0;
		for (int outputNeuronIdx = 0; outputNeuronIdx < numOutputNeurons; outputNeuronIdx++) 
		{
			sum += weightMatrix[(outputNeuronIdx * numInputNeurons) + neuronIdx] * input_errors[outputNeuronIdx];
		}
		double z = activations[neuronIdx];
		output_errors[neuronIdx] = sum * (exp(-z) / pow(1 + exp(-z), 2));
	}

	__global__ void AverageErrors(double* error, int mini) 
	{
		error[threadIdx.x] /= 15;
	}
}
int main()
{
	return 0;
}