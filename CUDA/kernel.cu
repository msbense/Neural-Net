#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <device_atomic_functions.h>
#include <device_atomic_functions.hpp>

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C"
{
	//Multiplies input vector by weight vector
	__global__ void FeedFoward(int *inputs, int *weightMatrix, int *activations, int numInputNeurons)
	{
		int neuronIdx = blockIdx.x;
		int inputNeuronIdx = threadIdx.x;
		__shared__ double sum;
		double result = weightMatrix[(neuronIdx * blockDim.x) + inputNeuronIdx] * inputs[inputNeuronIdx];
	}	
}

int main()
{
	return 0;
}