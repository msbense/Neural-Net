#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <device_atomic_functions.h>
#include <device_double_functions.h>
#include <math.h>

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C"
{
	//Multiplies input vector by weight vector
	__global__ void FeedFoward(double *inputs, double *weightMatrix, double* biases, double *activations, int numInputNeurons)
	{
		int neuronIdx = threadIdx.x;
		double sum = biases[neuronIdx];
		for (int inputNeuronIdx = 0; inputNeuronIdx < numInputNeurons; inputNeuronIdx++)
		{
			sum += weightMatrix[(neuronIdx * numInputNeurons) + inputNeuronIdx] * inputs[inputNeuronIdx];
		}
		activations[neuronIdx] = (1/(1+exp(-sum)));
	}

	__global__ void BackPropFirstLayer(double* errors, double* activations, double* correct) 
	{
		int neuronIdx = threadIdx.x;
		errors[neuronIdx] = activations[neuronIdx] - correct[neuronIdx];
	}

	__global__ void BackProp(double* input_errors, double* output_errors, double* activations, double* weightMatrix, int numOutputNeurons, int numInputNeurons)
	{
		int neuronIdx = threadIdx.x;
		double sum = 0;
		for (int outputNeuronIdx = 0; outputNeuronIdx < numOutputNeurons; outputNeuronIdx++) 
		{
			sum += weightMatrix[(outputNeuronIdx * numInputNeurons) + neuronIdx] * input_errors[outputNeuronIdx];
		}
		double z = activations[neuronIdx];
		output_errors[neuronIdx] = sum * (exp(-z) / pow(1 + exp(-z), 2));
	}

	__global__ void AverageAndCorrect(double* errors, double* weightMatrix, double* biases, double* prevLayerActivations, int numInputNeurons, int miniBatchSize) 
	{
		int neuronIdx = threadIdx.x;
		errors[neuronIdx] /= miniBatchSize;
		double error = errors[neuronIdx];
		biases[neuronIdx] = error;

		for (int inputNeuronIdx = 0; inputNeuronIdx < numInputNeurons; inputNeuronIdx++) 
		{
			weightMatrix[(neuronIdx * numInputNeurons) + inputNeuronIdx] = error * prevLayerActivations[inputNeuronIdx];
		}
	}
}
int main()
{
	return 0;
}